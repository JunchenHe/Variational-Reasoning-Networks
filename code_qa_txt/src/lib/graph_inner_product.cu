#include "hip/hip_runtime.h"
#include "graph_inner_product.h"
#include "tensor/gpu_handle.h"
#include "tensor/gpu_unary_functor.h"

namespace gnn
{

template<typename Dtype>
__global__ void SetValKernel(Dtype *dst, Dtype *src, int* entity_idx, int* sample_idx, int cols, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        dst[sample_idx[i] * cols + entity_idx[i]] = src[i];
    }
}

template<typename Dtype>
void SetVal(DTensor<GPU, Dtype>& src, int* entity_idx, int* sample_idx, DTensor<GPU, Dtype>& dst)
{
    int thread_num = c_uCudaThreadNum;
	if (src.shape.Count() < thread_num)
		thread_num = src.shape.Count();
    int blocksPerGrid = (src.shape.Count() + thread_num - 1) / thread_num;

    SetValKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>>(dst.data->ptr, src.data->ptr, entity_idx, sample_idx, dst.cols(), src.shape.Count());
}

template void SetVal(DTensor<GPU, float>& src, int* entity_idx, int* sample_idx, DTensor<GPU, float>& dst);
template void SetVal(DTensor<GPU, double>& src, int* entity_idx, int* sample_idx, DTensor<GPU, double>& dst);

template<typename Dtype>
__global__ void BpErrorKernel(Dtype *dst, Dtype *src, int* entity_idx, int* sample_idx, int cols, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        dst[i] = src[sample_idx[i] * cols + entity_idx[i]];
    }
}

template<typename Dtype>
void BpError(DTensor<GPU, Dtype>& grad_out, int* entity_idx, int* sample_idx, DTensor<GPU, Dtype>& cur_grad)
{
    int thread_num = c_uCudaThreadNum;
	if (cur_grad.shape.Count() < thread_num)
		thread_num = cur_grad.shape.Count();
    int blocksPerGrid = (cur_grad.shape.Count() + thread_num - 1) / thread_num;

    BpErrorKernel <<< blocksPerGrid, thread_num, 0, hipStreamPerThread >>>(cur_grad.data->ptr, grad_out.data->ptr, entity_idx, sample_idx, grad_out.cols(), cur_grad.shape.Count());
}

template void BpError(DTensor<GPU, float>& grad_out, int* entity_idx, int* sample_idx, DTensor<GPU, float>& cur_grad);
template void BpError(DTensor<GPU, double>& grad_out, int* entity_idx, int* sample_idx, DTensor<GPU, double>& cur_grad);

}